// nvcc -lineinfo -res-usage -arch=sm_75 -std=c++14 22111090-prob3-v3.cu -o 22111090-prob3-v3

#include <bits/stdc++.h>
#include <hip/hip_runtime.h>

using namespace std;

#define NSEC_SEC_MUL (1.0e9)

void gridloopsearch(double *a, double *b, double kk);

#define cudaCheckError(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

struct timespec begin_grid, end_main;

// to store values of disp.txt
double a[120];

// to store values of grid.txt
double b[30];

int main() {
  int i, j;

  i = 0;
  FILE* fp = fopen("./disp.txt", "r");
  if (fp == NULL) {
    printf("Error: could not open file\n");
    return 1;
  }

  while (!feof(fp)) {
    if (!fscanf(fp, "%lf", &a[i])) {
      printf("Error: fscanf failed while reading disp.txt\n");
      exit(EXIT_FAILURE);
    }
    i++;
  }
  fclose(fp);

  // read grid file
  j = 0;
  FILE* fpq = fopen("./grid.txt", "r");
  if (fpq == NULL) {
    printf("Error: could not open file\n");
    return 1;
  }

  while (!feof(fpq)) {
    if (!fscanf(fpq, "%lf", &b[j])) {
      printf("Error: fscanf failed while reading grid.txt\n");
      exit(EXIT_FAILURE);
    }
    j++;
  }
  fclose(fpq);

  // grid value initialize
  // initialize value of kk;
  double kk = 0.3;

  clock_gettime(CLOCK_MONOTONIC_RAW, &begin_grid);
  gridloopsearch(a, b, kk);
  clock_gettime(CLOCK_MONOTONIC_RAW, &end_main);
  printf("Total time = %f seconds\n", (end_main.tv_nsec - begin_grid.tv_nsec) / NSEC_SEC_MUL +
                                          (end_main.tv_sec - begin_grid.tv_sec));

  return EXIT_SUCCESS;
}

__global__ void gridloopsearch_kernel(int* pnts, double* results, double *a, double *b, double *kk) {

  extern __shared__ double shmem[][3];

  double dd1=b[0],dd2=b[1],dd3=b[2],dd4=b[3],dd5=b[4],dd6=b[5],dd7=b[6],dd8=b[7],dd9=b[8],dd10=b[9];
  double dd11=b[10],dd12=b[11],dd13=b[12],dd14=b[13],dd15=b[14],dd16=b[15],dd17=b[16],dd18=b[17],dd19=b[18],dd20=b[19];
  double dd21=b[20],dd22=b[21],dd23=b[22],dd24=b[23],dd25=b[24],dd26=b[25],dd27=b[26],dd28=b[27],dd29=b[28],dd30=b[29];
  double c11=a[0],c12=a[1],c13=a[2],c14=a[3],c15=a[4],c16=a[5],c17=a[6],c18=a[7],c19=a[8],c110=a[9],d1=a[10],ey1=a[11];
  double c21=a[12],c22=a[13],c23=a[14],c24=a[15],c25=a[16],c26=a[17],c27=a[18],c28=a[19],c29=a[20],c210=a[21],d2=a[22];
  double ey2=a[23],c31=a[24],c32=a[25],c33=a[26],c34=a[27],c35=a[28],c36=a[29],c37=a[30],c38=a[31],c39=a[32],c310=a[33];
  double d3=a[34],ey3=a[35],c41=a[36],c42=a[37],c43=a[38],c44=a[39],c45=a[40],c46=a[41],c47=a[42],c48=a[43],c49=a[44];
  double c410=a[45],d4=a[46],ey4=a[47],c51=a[48],c52=a[49],c53=a[50],c54=a[51],c55=a[52],c56=a[53],c57=a[54],c58=a[55];
  double c59=a[56],c510=a[57],d5=a[58],ey5=a[59],c61=a[60],c62=a[61],c63=a[62],c64=a[63],c65=a[64],c66=a[65],c67=a[66];
  double c68=a[67],c69=a[68],c610=a[69],d6=a[70],ey6=a[71],c71=a[72],c72=a[73],c73=a[74],c74=a[75],c75=a[76],c76=a[77];
  double c77=a[78],c78=a[79],c79=a[80],c710=a[81],d7=a[82],ey7=a[83],c81=a[84],c82=a[85],c83=a[86],c84=a[87],c85=a[88];
  double c86=a[89],c87=a[90],c88=a[91],c89=a[92],c810=a[93],d8=a[94],ey8=a[95],c91=a[96],c92=a[97],c93=a[98],c94=a[99];
  double c95=a[100],c96=a[101],c97=a[102],c98=a[103],c99=a[104],c910=a[105],d9=a[106],ey9=a[107],c101=a[108],c102=a[109];
  double c103=a[110],c104=a[111],c105=a[112],c106=a[113],c107=a[114],c108=a[115],c109=a[116],c1010=a[117],d10=a[118],ey10=a[119];

  int r1 = blockIdx.x * blockDim.x + threadIdx.x;
  int r2 = blockIdx.y * blockDim.y + threadIdx.y;
  int r3 = blockIdx.z * blockDim.z + threadIdx.z;
  int tidx = threadIdx.x, tidy = threadIdx.y, tidz = threadIdx.z;

  int s1, s2, s3, s4, s5, s6, s7, s8, s9, s10;
  s1 = static_cast<int>(floor((dd2 - dd1) / dd3));
  s2 = static_cast<int>(floor((dd5 - dd4) / dd6));
  s3 = static_cast<int>(floor((dd8 - dd7) / dd9));
  s4 = static_cast<int>(floor((dd11 - dd10) / dd12));
  s5 = static_cast<int>(floor((dd14 - dd13) / dd15));
  s6 = static_cast<int>(floor((dd17 - dd16) / dd18));
  s7 = static_cast<int>(floor((dd20 - dd19) / dd21));
  s8 = static_cast<int>(floor((dd23 - dd22) / dd24));
  s9 = static_cast<int>(floor((dd26 - dd25) / dd27));
  s10 = static_cast<int>(floor((dd29 - dd28) / dd30));

  double e1, e2, e3, e4, e5, e6, e7, e8, e9, e10;
  e1 = (*kk) * ey1;
  e2 = (*kk) * ey2;
  e3 = (*kk) * ey3;
  e4 = (*kk) * ey4;
  e5 = (*kk) * ey5;
  e6 = (*kk) * ey6;
  e7 = (*kk) * ey7;
  e8 = (*kk) * ey8;
  e9 = (*kk) * ey9;
  e10 = (*kk) * ey10;

  double x1, x2, x3, x4, x5, x6, x7, x8, x9, x10;
  double q1, q2, q3, q4, q5, q6, q7, q8, q9, q10;

  int shmem_idx = tidx + blockDim.x*(tidy + blockDim.y*tidz);

  if (r1 < s1 && r2 < s2 && r3 < s3) {
    shmem[shmem_idx][0] = dd1 + r1*dd3;
    shmem[shmem_idx][1] = dd4 + r2*dd6;
    shmem[shmem_idx][2] = dd7 + r3*dd9;
  }

  __syncthreads();

  if (r1 < s1 && r2 < s2 && r3 < s3) {

    for (int r4 = 0; r4 < s4; r4++) {
      x4 = dd10 + r4 * dd12;

      for (int r5 = 0; r5 < s5; r5++) {
        x5 = dd13 + r5 * dd15;

        for (int r6 = 0; r6 < s6; r6++) {
          x6 = dd16 + r6 * dd18;

          for (int r7 = 0; r7 < s7; r7++) {
            x7 = dd19 + r7 * dd21;

            for (int r8 = 0; r8 < s8; r8++) {
              x8 = dd22 + r8 * dd24;

              for (int r9 = 0; r9 < s9; r9++) {
                x9 = dd25 + r9 * dd27;

                for (int r10 = 0; r10 < s10; r10++) {
                  x10 = dd28 + r10 * dd30;
                  x1 = shmem[shmem_idx][0];
                  x2 = shmem[shmem_idx][1];
                  x3 = shmem[shmem_idx][2];

                  // constraints

                  q1 = fabs(c11 * x1 + c12 * x2 + c13 * x3 + c14 * x4 + c15 * x5 + c16 * x6 +
                            c17 * x7 + c18 * x8 + c19 * x9 + c110 * x10 - d1);

                  q2 = fabs(c21 * x1 + c22 * x2 + c23 * x3 + c24 * x4 + c25 * x5 + c26 * x6 +
                          c27 * x7 + c28 * x8 + c29 * x9 + c210 * x10 - d2);

                  q3 = fabs(c31 * x1 + c32 * x2 + c33 * x3 + c34 * x4 + c35 * x5 + c36 * x6 +
                          c37 * x7 + c38 * x8 + c39 * x9 + c310 * x10 - d3);

                  q4 = fabs(c41 * x1 + c42 * x2 + c43 * x3 + c44 * x4 + c45 * x5 + c46 * x6 +
                          c47 * x7 + c48 * x8 + c49 * x9 + c410 * x10 - d4);

                  q5 = fabs(c51 * x1 + c52 * x2 + c53 * x3 + c54 * x4 + c55 * x5 + c56 * x6 +
                          c57 * x7 + c58 * x8 + c59 * x9 + c510 * x10 - d5);

                  q6 = fabs(c61 * x1 + c62 * x2 + c63 * x3 + c64 * x4 + c65 * x5 + c66 * x6 +
                          c67 * x7 + c68 * x8 + c69 * x9 + c610 * x10 - d6);

                  q7 = fabs(c71 * x1 + c72 * x2 + c73 * x3 + c74 * x4 + c75 * x5 + c76 * x6 +
                          c77 * x7 + c78 * x8 + c79 * x9 + c710 * x10 - d7);

                  q8 = fabs(c81 * x1 + c82 * x2 + c83 * x3 + c84 * x4 + c85 * x5 + c86 * x6 +
                          c87 * x7 + c88 * x8 + c89 * x9 + c810 * x10 - d8);

                  q9 = fabs(c91 * x1 + c92 * x2 + c93 * x3 + c94 * x4 + c95 * x5 + c96 * x6 +
                          c97 * x7 + c98 * x8 + c99 * x9 + c910 * x10 - d9);

                  q10 = fabs(c101 * x1 + c102 * x2 + c103 * x3 + c104 * x4 + c105 * x5 +
                           c106 * x6 + c107 * x7 + c108 * x8 + c109 * x9 + c1010 * x10 - d10);

                  if ((q1 <= e1) && (q2 <= e2) && (q3 <= e3) && (q4 <= e4) && (q5 <= e5) &&
                    (q6 <= e6) && (q7 <= e7) && (q8 <= e8) && (q9 <= e9) && (q10 <= e10)) {
                      int i = atomicAdd(pnts, 1);
                      results[i * 10] = x1;
                      results[i * 10 + 1] = x2;
                      results[i * 10 + 2] = x3;
                      results[i * 10 + 3] = x4;
                      results[i * 10 + 4] = x5;
                      results[i * 10 + 5] = x6;
                      results[i * 10 + 6] = x7;
                      results[i * 10 + 7] = x8;
                      results[i * 10 + 8] = x9;
                      results[i * 10 + 9] = x10;
                  }
                }
              }
            }
          }
        }
      }
    }
  }
}

void gridloopsearch(double *a, double *b, double kk) {
    FILE* fptr = fopen("./results-v3.txt", "w");
    if (fptr == NULL) {
      printf("Error in creating file !");
      exit(1);
    }

    int device;
    hipGetDevice(&device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    int sharedMemoryPerBlock = prop.sharedMemPerBlock;

    double *h_results, *h_kk, *h_a, *h_b;
    const int max_results = 20000;
    h_results = new double[max_results*10];

    int *h_pnts;

    cudaCheckError(hipMallocManaged((void**)&h_a, 120 * sizeof(double)));
    cudaCheckError(hipMallocManaged((void**)&h_b, 30 * sizeof(double)));
    cudaCheckError(hipMallocManaged((void**)&h_kk, 1 * sizeof(double)));
    cudaCheckError(hipMallocManaged((void**)&h_results, max_results * 10 * sizeof(double)));
    cudaCheckError(hipMallocManaged((void**)&h_pnts, 1 * sizeof(int)));

    cudaCheckError(hipMemcpy(h_a, a, 120 * sizeof(double), hipMemcpyHostToDevice));
    cudaCheckError(hipMemcpy(h_b, b, 30 * sizeof(double), hipMemcpyHostToDevice));

    *h_kk = kk;
    *h_pnts = 0;

    // Launch the kernel

    int s1, s2, s3;
    s1 = static_cast<int>(floor((b[1] - b[0]) / b[2]));
    s2 = static_cast<int>(floor((b[4] - b[3]) / b[5]));
    s3 = static_cast<int>(floor((b[7] - b[6]) / b[8]));

    int x = 4, y = 4, z = 4;
    dim3 gridSize((s1+x-1)/x, (s2+y-1)/y, (s3+z-1)/z);
    dim3 blockSize(x, y, z);

    int sz = blockSize.x + blockSize.x*(blockSize.y + blockSize.y*blockSize.z);

    assert((sz * 3 * sizeof(double)) <= sharedMemoryPerBlock);

    hipEvent_t start, end;
    float time_elapsed;
    hipEventCreate(&start);
    hipEventCreate(&end);

    hipEventRecord(start);
    gridloopsearch_kernel<<<gridSize, blockSize, sz * 3 * sizeof(double)>>>(h_pnts, h_results, h_a, h_b, h_kk);
    cudaCheckError(hipPeekAtLastError());
    cudaCheckError(hipDeviceSynchronize());
    hipEventRecord(end);
    hipEventSynchronize(end);
    hipEventElapsedTime(&time_elapsed, start, end);

    cout << "CUDA time elapsed: " << time_elapsed / 1000.0 << " seconds" << endl;

    vector <vector <double>> arr;

    for (int i = 0; i < *h_pnts; i++) {
        vector <double> row;
        for (int j = 0; j < 10; j++) row.push_back(h_results[i*10 + j]);
        arr.push_back(row);
    }

    sort(arr.begin(), arr.end());

    for (int i = 0; i < arr.size(); i++) {
        fprintf(fptr, "%lf\t", arr[i][0]);
        fprintf(fptr, "%lf\t", arr[i][1]);
        fprintf(fptr, "%lf\t", arr[i][2]);
        fprintf(fptr, "%lf\t", arr[i][3]);
        fprintf(fptr, "%lf\t", arr[i][4]);
        fprintf(fptr, "%lf\t", arr[i][5]);
        fprintf(fptr, "%lf\t", arr[i][6]);
        fprintf(fptr, "%lf\t", arr[i][7]);
        fprintf(fptr, "%lf\t", arr[i][8]);
        fprintf(fptr, "%lf\n", arr[i][9]);
    }

    fclose(fptr);
    printf("result pnts: %d\n", *h_pnts);

    hipFree(h_pnts);
    hipFree(h_results);
    hipFree(h_kk);
    hipFree(h_a);
    hipFree(h_b);
}